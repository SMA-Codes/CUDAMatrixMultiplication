#include "hip/hip_runtime.h"
#include "GPUMatrixMultiplication.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 16 // Ensure this is a supported size for your GPU

// CUDA Kernel
__global__ void MatrixMultiplicationKernel(const float* matrixA, const float* matrixB, float* matrixC, int matrixDimension) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < matrixDimension && col < matrixDimension) {
        float sum = 0.0f;
        for (int i = 0; i < matrixDimension; i++) {
            sum += matrixA[row * matrixDimension + i] * matrixB[i * matrixDimension + col];
        }
        matrixC[row * matrixDimension + col] = sum;
    }
}


// Host Function
auto GPUMatrixMultiplication::SquareMatrixMultiplication(const float* matrixA, const float* matrixB, float* matrixC, int matrixDimension) -> void {
    float *d_matrixA, *d_matrixB, *d_matrixC;

    size_t matrixSize = matrixDimension * matrixDimension * sizeof(float);

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    if (3 * matrixSize > freeMem) {
        std::cerr << "Insufficient GPU memory for this matrix size!" << std::endl;
        return;
    }

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_matrixA, matrixSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_matrixA: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc((void**)&d_matrixB, matrixSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_matrixB: " << hipGetErrorString(err) << std::endl;
        hipFree(d_matrixA);
        return;
    }

    err = hipMalloc((void**)&d_matrixC, matrixSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_matrixC: " << hipGetErrorString(err) << std::endl;
        hipFree(d_matrixA);
        hipFree(d_matrixB);
        return;
    }

    // Copy host data to device
    err = hipMemcpy(d_matrixA, matrixA, matrixSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_matrixA: " << hipGetErrorString(err) << std::endl;
        hipFree(d_matrixA);
        hipFree(d_matrixB);
        hipFree(d_matrixC);
        return;
    }

    err = hipMemcpy(d_matrixB, matrixB, matrixSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_matrixB: " << hipGetErrorString(err) << std::endl;
        hipFree(d_matrixA);
        hipFree(d_matrixB);
        hipFree(d_matrixC);
        return;
    }

    // Define thread block and grid dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dimGrid((matrixDimension + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                 (matrixDimension + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                 1);

    // Launch kernel
    MatrixMultiplicationKernel<<<dimGrid, dimBlock>>>(d_matrixA, d_matrixB, d_matrixC, matrixDimension);

    // Synchronize and check for errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_matrixA);
        hipFree(d_matrixB);
        hipFree(d_matrixC);
        return;
    }

    // Copy result back to host
    err = hipMemcpy(matrixC, d_matrixC, matrixSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for d_matrixC: " << hipGetErrorString(err) << std::endl;
    }

    // Free device memory
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
}
